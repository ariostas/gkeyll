#include "hip/hip_runtime.h"
/* -*- c++ -*- */
extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array.h>
#include <gkyl_array_ops.h>
#include <gkyl_basis.h>
#include <gkyl_math.h>
#include <gkyl_util.h>
#include <gkyl_gk_geometry.h>
#include <assert.h>
}

__global__ static void
gk_geometry_set_corn_cu_kernel(struct gk_geometry *gk_geom,
  struct gkyl_array *mc2p, struct gkyl_array *mc2nu_pos, struct gkyl_array *bmag,
  struct gkyl_array *mc2p_nodal, struct gkyl_array *mc2nu_pos_nodal, struct gkyl_array *bmag_nodal 
 )
{
  // Expansions.
  gk_geom->geo_corn.mc2p = mc2p;
  gk_geom->geo_corn.mc2nu_pos = mc2nu_pos;
  gk_geom->geo_corn.bmag = bmag;
  // Nodal.
  gk_geom->geo_corn.mc2p_nodal = mc2p_nodal;
  gk_geom->geo_corn.mc2nu_pos_nodal = mc2nu_pos_nodal;
  gk_geom->geo_corn.bmag_nodal = bmag_nodal;
}

__global__ static void
gk_geometry_set_int_cu_kernel(struct gk_geometry *gk_geom,
  struct gkyl_array *mc2p, struct gkyl_array *bmag, struct gkyl_array *g_ij, struct gkyl_array *g_ij_neut,
  struct gkyl_array *dxdz, struct gkyl_array *dzdx, struct gkyl_array *dualmag, struct gkyl_array *normals,
  struct gkyl_array *jacobgeo, struct gkyl_array *jacobgeo_ghost, struct gkyl_array *jacobgeo_inv, struct gkyl_array *gij,
  struct gkyl_array *gij_neut, struct gkyl_array *b_i, struct gkyl_array *bcart, struct gkyl_array *cmag,
  struct gkyl_array *jacobtot, struct gkyl_array *jacobtot_inv, struct gkyl_array *bmag_inv, struct gkyl_array *bmag_inv_sq,
  struct gkyl_array *gxxj, struct gkyl_array *gxyj, struct gkyl_array *gyyj, struct gkyl_array *gxzj,
  struct gkyl_array *eps2, struct gkyl_array *rtg33inv, struct gkyl_array *dualcurlbhatoverB, struct gkyl_array *bioverJB,
  struct gkyl_array *B3, struct gkyl_array *dualcurlbhat, struct gkyl_array *mc2p_nodal_fd, struct gkyl_array *mc2p_nodal,
  struct gkyl_array *bmag_nodal, struct gkyl_array *ddtheta_nodal, struct gkyl_array *curlbhat_nodal,
  struct gkyl_array *dualcurlbhat_nodal, struct gkyl_array *jacobgeo_nodal, struct gkyl_array *g_ij_nodal,
  struct gkyl_array *g_ij_neut_nodal, struct gkyl_array *dxdz_nodal, struct gkyl_array *dzdx_nodal,
  struct gkyl_array *dualmag_nodal, struct gkyl_array *normals_nodal, struct gkyl_array *gij_neut_nodal,
  struct gkyl_array *b_i_nodal, struct gkyl_array *b_i_nodal_fd, struct gkyl_array *bcart_nodal,
  struct gkyl_array *B3_nodal, struct gkyl_array *dualcurlbhatoverB_nodal, struct gkyl_array *rtg33inv_nodal,
  struct gkyl_array *bioverJB_nodal 
 )
{
  // Expansions.
  gk_geom->geo_int.mc2p = mc2p;
  gk_geom->geo_int.bmag = bmag;
  gk_geom->geo_int.g_ij = g_ij;
  gk_geom->geo_int.g_ij_neut = g_ij_neut;
  gk_geom->geo_int.dxdz = dxdz;
  gk_geom->geo_int.dzdx = dzdx;
  gk_geom->geo_int.dualmag = dualmag;
  gk_geom->geo_int.normals = normals;
  gk_geom->geo_int.jacobgeo = jacobgeo;
  gk_geom->geo_int.jacobgeo_ghost = jacobgeo_ghost;
  gk_geom->geo_int.jacobgeo_inv = jacobgeo_inv;
  gk_geom->geo_int.gij = gij;
  gk_geom->geo_int.gij_neut = gij_neut;
  gk_geom->geo_int.b_i = b_i;
  gk_geom->geo_int.bcart = bcart;
  gk_geom->geo_int.cmag = cmag;
  gk_geom->geo_int.jacobtot = jacobtot;
  gk_geom->geo_int.jacobtot_inv = jacobtot_inv;
  gk_geom->geo_int.bmag_inv = bmag_inv;
  gk_geom->geo_int.bmag_inv_sq = bmag_inv_sq;
  gk_geom->geo_int.gxxj = gxxj;
  gk_geom->geo_int.gxyj = gxyj;
  gk_geom->geo_int.gyyj = gyyj;
  gk_geom->geo_int.gxzj = gxzj;
  gk_geom->geo_int.eps2 = eps2;
  gk_geom->geo_int.rtg33inv = rtg33inv;
  gk_geom->geo_int.dualcurlbhatoverB = dualcurlbhatoverB;
  gk_geom->geo_int.dualcurlbhat = dualcurlbhat;
  gk_geom->geo_int.bioverJB = bioverJB;
  gk_geom->geo_int.B3 = B3;
  // Nodal.
  gk_geom->geo_int.mc2p_nodal_fd = mc2p_nodal_fd;
  gk_geom->geo_int.mc2p_nodal = mc2p_nodal;
  gk_geom->geo_int.bmag_nodal = bmag_nodal;
  gk_geom->geo_int.ddtheta_nodal = ddtheta_nodal;
  gk_geom->geo_int.curlbhat_nodal = curlbhat_nodal;
  gk_geom->geo_int.dualcurlbhat_nodal = dualcurlbhat_nodal;
  gk_geom->geo_int.jacobgeo_nodal = jacobgeo_nodal;
  gk_geom->geo_int.g_ij_nodal = g_ij_nodal;
  gk_geom->geo_int.g_ij_neut_nodal = g_ij_neut_nodal;
  gk_geom->geo_int.dxdz_nodal = dxdz_nodal;
  gk_geom->geo_int.dzdx_nodal = dzdx_nodal;
  gk_geom->geo_int.dualmag_nodal = dualmag_nodal;
  gk_geom->geo_int.normals_nodal = normals_nodal;
  gk_geom->geo_int.gij_neut_nodal = gij_neut_nodal;
  gk_geom->geo_int.b_i_nodal = b_i_nodal;
  gk_geom->geo_int.b_i_nodal_fd = b_i_nodal_fd;
  gk_geom->geo_int.bcart_nodal = bcart_nodal;
  gk_geom->geo_int.B3_nodal = B3_nodal;
  gk_geom->geo_int.dualcurlbhatoverB_nodal = dualcurlbhatoverB_nodal;
  gk_geom->geo_int.rtg33inv_nodal = rtg33inv_nodal;
  gk_geom->geo_int.bioverJB_nodal = bioverJB_nodal;
}

__global__ static void
gk_geometry_set_surf_cu_kernel(struct gk_geometry *gk_geom, int dir,
  struct gkyl_array *bmag, struct gkyl_array *jacobgeo, struct gkyl_array *jacobgeo_sync, 
  struct gkyl_array *b_i, struct gkyl_array *cmag, struct gkyl_array *jacobtot_inv,
  struct gkyl_array *B3, struct gkyl_array *normcurlbhat, struct gkyl_array *normals, struct gkyl_array *lenr,
  struct gkyl_array *mc2p_nodal_fd, struct gkyl_array *mc2p_nodal, struct gkyl_array *bmag_nodal,
  struct gkyl_array *curlbhat_nodal, struct gkyl_array *normcurlbhat_nodal, struct gkyl_array *ddtheta_nodal,
  struct gkyl_array *jacobgeo_nodal, struct gkyl_array *b_i_nodal, struct gkyl_array *b_i_nodal_fd,
  struct gkyl_array *cmag_nodal, struct gkyl_array *jacobtot_inv_nodal, struct gkyl_array *g_ij_nodal,
  struct gkyl_array *dxdz_nodal, struct gkyl_array *dzdx_nodal, struct gkyl_array *normals_nodal,
  struct gkyl_array *dualmag_nodal, struct gkyl_array *bcart_nodal, struct gkyl_array *B3_nodal,
  struct gkyl_array *lenr_nodal 
 )
{
  // Expansions.
  gk_geom->geo_surf[dir].bmag = bmag;
  gk_geom->geo_surf[dir].jacobgeo = jacobgeo;
  gk_geom->geo_surf[dir].jacobgeo_sync = jacobgeo_sync;
  gk_geom->geo_surf[dir].b_i = b_i;
  gk_geom->geo_surf[dir].cmag = cmag;
  gk_geom->geo_surf[dir].jacobtot_inv = jacobtot_inv;
  gk_geom->geo_surf[dir].B3 = B3;
  gk_geom->geo_surf[dir].normcurlbhat = normcurlbhat;
  gk_geom->geo_surf[dir].normals = normals;
  gk_geom->geo_surf[dir].lenr = lenr;
  // Nodal.
  gk_geom->geo_surf[dir].mc2p_nodal_fd = mc2p_nodal_fd;
  gk_geom->geo_surf[dir].mc2p_nodal = mc2p_nodal;
  gk_geom->geo_surf[dir].bmag_nodal = bmag_nodal;
  gk_geom->geo_surf[dir].curlbhat_nodal = curlbhat_nodal;
  gk_geom->geo_surf[dir].normcurlbhat_nodal = normcurlbhat_nodal;
  gk_geom->geo_surf[dir].ddtheta_nodal = ddtheta_nodal;
  gk_geom->geo_surf[dir].jacobgeo_nodal = jacobgeo_nodal;
  gk_geom->geo_surf[dir].b_i_nodal = b_i_nodal;
  gk_geom->geo_surf[dir].b_i_nodal_fd = b_i_nodal_fd;
  gk_geom->geo_surf[dir].cmag_nodal = cmag_nodal;
  gk_geom->geo_surf[dir].jacobtot_inv_nodal = jacobtot_inv_nodal;
  gk_geom->geo_surf[dir].g_ij_nodal = g_ij_nodal;
  gk_geom->geo_surf[dir].dxdz_nodal = dxdz_nodal;
  gk_geom->geo_surf[dir].dzdx_nodal = dzdx_nodal;
  gk_geom->geo_surf[dir].normals_nodal = normals_nodal;
  gk_geom->geo_surf[dir].dualmag_nodal = dualmag_nodal;
  gk_geom->geo_surf[dir].bcart_nodal = bcart_nodal;
  gk_geom->geo_surf[dir].B3_nodal = B3_nodal;
  gk_geom->geo_surf[dir].lenr_nodal = lenr_nodal;
}

// Host-side wrapper for set_corn_cu_kernel
void
gkyl_geometry_set_corn_cu(struct gk_geometry *gk_geom, struct gk_geom_corn *geo_corn)
{
  gk_geometry_set_corn_cu_kernel<<<1,1>>>(gk_geom, 
    geo_corn->mc2p->on_dev, geo_corn->mc2nu_pos->on_dev, geo_corn->bmag->on_dev,
    geo_corn->mc2p_nodal->on_dev, geo_corn->mc2nu_pos_nodal->on_dev, geo_corn->bmag_nodal->on_dev);
}

// Host-side wrapper for set_int_cu_kernel
void
gkyl_geometry_set_int_cu(struct gk_geometry *gk_geom, struct gk_geom_int *geo_int)
{
  gk_geometry_set_int_cu_kernel<<<1,1>>>(gk_geom,
    geo_int->mc2p->on_dev, geo_int->bmag->on_dev, geo_int->g_ij->on_dev, geo_int->g_ij_neut->on_dev,
    geo_int->dxdz->on_dev, geo_int->dzdx->on_dev, geo_int->dualmag->on_dev, geo_int->normals->on_dev,
    geo_int->jacobgeo->on_dev, geo_int->jacobgeo_ghost->on_dev, geo_int->jacobgeo_inv->on_dev, geo_int->gij->on_dev,
    geo_int->gij_neut->on_dev, geo_int->b_i->on_dev, geo_int->bcart->on_dev, geo_int->cmag->on_dev,
    geo_int->jacobtot->on_dev, geo_int->jacobtot_inv->on_dev, geo_int->bmag_inv->on_dev, geo_int->bmag_inv_sq->on_dev,
    geo_int->gxxj->on_dev, geo_int->gxyj->on_dev, geo_int->gyyj->on_dev, geo_int->gxzj->on_dev,
    geo_int->eps2->on_dev, geo_int->rtg33inv->on_dev, geo_int->dualcurlbhatoverB->on_dev, geo_int->bioverJB->on_dev,
    geo_int->B3->on_dev, geo_int->dualcurlbhat->on_dev,
    geo_int->mc2p_nodal_fd->on_dev, geo_int->mc2p_nodal->on_dev, geo_int->bmag_nodal->on_dev,
    geo_int->ddtheta_nodal->on_dev, geo_int->curlbhat_nodal->on_dev, geo_int->dualcurlbhat_nodal->on_dev,
    geo_int->jacobgeo_nodal->on_dev, geo_int->g_ij_nodal->on_dev, geo_int->g_ij_neut_nodal->on_dev,
    geo_int->dxdz_nodal->on_dev, geo_int->dzdx_nodal->on_dev, geo_int->dualmag_nodal->on_dev,
    geo_int->normals_nodal->on_dev, geo_int->gij_neut_nodal->on_dev, geo_int->b_i_nodal->on_dev,
    geo_int->b_i_nodal_fd->on_dev, geo_int->bcart_nodal->on_dev, geo_int->B3_nodal->on_dev,
    geo_int->dualcurlbhatoverB_nodal->on_dev, geo_int->rtg33inv_nodal->on_dev, geo_int->bioverJB_nodal->on_dev);
}

// Host-side wrapper for set_surf_cu_kernel
void
gkyl_geometry_set_surf_cu(struct gk_geometry *gk_geom, struct gk_geom_surf *geo_surf, int dir)
{
 gk_geometry_set_surf_cu_kernel<<<1,1>>>(gk_geom, dir,
   geo_surf->bmag->on_dev, geo_surf->jacobgeo->on_dev, geo_surf->jacobgeo_sync->on_dev, 
   geo_surf->b_i->on_dev, geo_surf->cmag->on_dev, geo_surf->jacobtot_inv->on_dev, geo_surf->B3->on_dev,
   geo_surf->normcurlbhat->on_dev, geo_surf->normals->on_dev, geo_surf->lenr->on_dev,
   geo_surf->mc2p_nodal_fd->on_dev, geo_surf->mc2p_nodal->on_dev, geo_surf->bmag_nodal->on_dev,
   geo_surf->curlbhat_nodal->on_dev, geo_surf->normcurlbhat_nodal->on_dev, geo_surf->ddtheta_nodal->on_dev,
   geo_surf->jacobgeo_nodal->on_dev, geo_surf->b_i_nodal->on_dev, geo_surf->b_i_nodal_fd->on_dev,
   geo_surf->cmag_nodal->on_dev, geo_surf->jacobtot_inv_nodal->on_dev, geo_surf->g_ij_nodal->on_dev,
   geo_surf->dxdz_nodal->on_dev, geo_surf->dzdx_nodal->on_dev, geo_surf->normals_nodal->on_dev,
   geo_surf->dualmag_nodal->on_dev, geo_surf->bcart_nodal->on_dev, geo_surf->B3_nodal->on_dev,
   geo_surf->lenr_nodal->on_dev);
}

struct gk_geom_corn*
gk_geometry_corn_cu_dev_alloc(struct gk_geom_corn up_corn_host)
{
  struct gk_geom_corn *up_corn_dev = (struct gk_geom_corn*) gkyl_malloc(sizeof(struct gk_geom_corn));
  // Expansions.
  up_corn_dev->mc2p = gkyl_array_cu_dev_new(up_corn_host.mc2p->type,
  up_corn_host.mc2p->ncomp, up_corn_host.mc2p->size);
  up_corn_dev->mc2nu_pos = gkyl_array_cu_dev_new(up_corn_host.mc2nu_pos->type,
  up_corn_host.mc2nu_pos->ncomp, up_corn_host.mc2nu_pos->size);
  up_corn_dev->bmag = gkyl_array_cu_dev_new(up_corn_host.bmag->type,
  up_corn_host.bmag->ncomp, up_corn_host.bmag->size);
  up_corn_dev->mc2p_deflated = gkyl_array_cu_dev_new(up_corn_host.mc2p_deflated->type,
  up_corn_host.mc2p_deflated->ncomp, up_corn_host.mc2p_deflated->size);
  up_corn_dev->mc2nu_pos_deflated = gkyl_array_cu_dev_new(up_corn_host.mc2nu_pos_deflated->type,
  up_corn_host.mc2nu_pos_deflated->ncomp, up_corn_host.mc2nu_pos_deflated->size);
  // Nodal.
  up_corn_dev->mc2p_nodal = gkyl_array_cu_dev_new(up_corn_host.mc2p_nodal->type,
  up_corn_host.mc2p_nodal->ncomp, up_corn_host.mc2p_nodal->size);
  up_corn_dev->mc2nu_pos_nodal = gkyl_array_cu_dev_new(up_corn_host.mc2nu_pos_nodal->type,
  up_corn_host.mc2nu_pos_nodal->ncomp, up_corn_host.mc2nu_pos_nodal->size);
  up_corn_dev->bmag_nodal = gkyl_array_cu_dev_new(up_corn_host.bmag_nodal->type,
  up_corn_host.bmag_nodal->ncomp, up_corn_host.bmag_nodal->size);
  return up_corn_dev;
}

struct gk_geom_int*
gk_geometry_int_cu_dev_alloc(struct gk_geom_int up_int_host)
{
  struct gk_geom_int *up_int_dev = (struct gk_geom_int*) gkyl_malloc(sizeof(struct gk_geom_int));
  // Expansions.
  up_int_dev->mc2p = gkyl_array_cu_dev_new(up_int_host.mc2p->type,
    up_int_host.mc2p->ncomp, up_int_host.mc2p->size);
  up_int_dev->bmag = gkyl_array_cu_dev_new(up_int_host.bmag->type,
    up_int_host.bmag->ncomp, up_int_host.bmag->size);
  up_int_dev->g_ij = gkyl_array_cu_dev_new(up_int_host.g_ij->type,
    up_int_host.g_ij->ncomp, up_int_host.g_ij->size);
  up_int_dev->g_ij_neut = gkyl_array_cu_dev_new(up_int_host.g_ij_neut->type,
    up_int_host.g_ij_neut->ncomp, up_int_host.g_ij_neut->size);
  up_int_dev->dxdz = gkyl_array_cu_dev_new(up_int_host.dxdz->type,
    up_int_host.dxdz->ncomp, up_int_host.dxdz->size);
  up_int_dev->dzdx = gkyl_array_cu_dev_new(up_int_host.dzdx->type,
    up_int_host.dzdx->ncomp, up_int_host.dzdx->size);
  up_int_dev->dualmag = gkyl_array_cu_dev_new(up_int_host.dualmag->type,
    up_int_host.dualmag->ncomp, up_int_host.dualmag->size);
  up_int_dev->normals = gkyl_array_cu_dev_new(up_int_host.normals->type,
    up_int_host.normals->ncomp, up_int_host.normals->size);
  up_int_dev->jacobgeo = gkyl_array_cu_dev_new(up_int_host.jacobgeo->type,
    up_int_host.jacobgeo->ncomp, up_int_host.jacobgeo->size);
  up_int_dev->jacobgeo_ghost = gkyl_array_cu_dev_new(up_int_host.jacobgeo_ghost->type,
    up_int_host.jacobgeo_ghost->ncomp, up_int_host.jacobgeo_ghost->size);
  up_int_dev->jacobgeo_inv = gkyl_array_cu_dev_new(up_int_host.jacobgeo_inv->type,
    up_int_host.jacobgeo_inv->ncomp, up_int_host.jacobgeo_inv->size);
  up_int_dev->gij = gkyl_array_cu_dev_new(up_int_host.gij->type,
    up_int_host.gij->ncomp, up_int_host.gij->size);
  up_int_dev->gij_neut = gkyl_array_cu_dev_new(up_int_host.gij_neut->type,
    up_int_host.gij_neut->ncomp, up_int_host.gij_neut->size);
  up_int_dev->b_i = gkyl_array_cu_dev_new(up_int_host.b_i->type,
    up_int_host.b_i->ncomp, up_int_host.b_i->size);
  up_int_dev->bcart = gkyl_array_cu_dev_new(up_int_host.bcart->type,
    up_int_host.bcart->ncomp, up_int_host.bcart->size);
  up_int_dev->cmag = gkyl_array_cu_dev_new(up_int_host.cmag->type, 
    up_int_host.cmag->ncomp, up_int_host.cmag->size);
  up_int_dev->jacobtot = gkyl_array_cu_dev_new(up_int_host.jacobtot->type,
    up_int_host.jacobtot->ncomp, up_int_host.jacobtot->size);
  up_int_dev->jacobtot_inv = gkyl_array_cu_dev_new(up_int_host.jacobtot_inv->type,
    up_int_host.jacobtot_inv->ncomp, up_int_host.jacobtot_inv->size);
  up_int_dev->bmag_inv = gkyl_array_cu_dev_new(up_int_host.bmag_inv->type,
    up_int_host.bmag_inv->ncomp, up_int_host.bmag_inv->size);
  up_int_dev->bmag_inv_sq = gkyl_array_cu_dev_new(up_int_host.bmag_inv_sq->type,
    up_int_host.bmag_inv_sq->ncomp, up_int_host.bmag_inv_sq->size);
  up_int_dev->gxxj = gkyl_array_cu_dev_new(up_int_host.gxxj->type, 
    up_int_host.gxxj->ncomp, up_int_host.gxxj->size);
  up_int_dev->gxyj = gkyl_array_cu_dev_new(up_int_host.gxyj->type, 
    up_int_host.gxyj->ncomp, up_int_host.gxyj->size);
  up_int_dev->gyyj = gkyl_array_cu_dev_new(up_int_host.gyyj->type, 
    up_int_host.gyyj->ncomp, up_int_host.gyyj->size);
  up_int_dev->gxzj = gkyl_array_cu_dev_new(up_int_host.gxzj->type, 
    up_int_host.gxzj->ncomp, up_int_host.gxzj->size);
  up_int_dev->eps2 = gkyl_array_cu_dev_new(up_int_host.eps2->type, 
    up_int_host.eps2->ncomp, up_int_host.eps2->size);
  up_int_dev->rtg33inv = gkyl_array_cu_dev_new(up_int_host.rtg33inv->type, 
    up_int_host.rtg33inv->ncomp, up_int_host.rtg33inv->size);
  up_int_dev->dualcurlbhatoverB = gkyl_array_cu_dev_new(up_int_host.dualcurlbhatoverB->type,
    up_int_host.dualcurlbhatoverB->ncomp, up_int_host.dualcurlbhatoverB->size);
  up_int_dev->bioverJB = gkyl_array_cu_dev_new(up_int_host.bioverJB->type, 
    up_int_host.bioverJB->ncomp, up_int_host.bioverJB->size);
  up_int_dev->B3 = gkyl_array_cu_dev_new(up_int_host.B3->type, 
    up_int_host.B3->ncomp, up_int_host.B3->size);
  up_int_dev->dualcurlbhat = gkyl_array_cu_dev_new(up_int_host.dualcurlbhat->type,
    up_int_host.dualcurlbhat->ncomp, up_int_host.dualcurlbhat->size);
  // Nodal.
  up_int_dev->mc2p_nodal_fd = gkyl_array_cu_dev_new(up_int_host.mc2p_nodal_fd->type,
    up_int_host.mc2p_nodal_fd->ncomp, up_int_host.mc2p_nodal_fd->size);
  up_int_dev->mc2p_nodal = gkyl_array_cu_dev_new(up_int_host.mc2p_nodal->type,
    up_int_host.mc2p_nodal->ncomp, up_int_host.mc2p_nodal->size);
  up_int_dev->bmag_nodal = gkyl_array_cu_dev_new(up_int_host.bmag_nodal->type,
    up_int_host.bmag_nodal->ncomp, up_int_host.bmag_nodal->size);
  up_int_dev->ddtheta_nodal = gkyl_array_cu_dev_new(up_int_host.ddtheta_nodal->type,
    up_int_host.ddtheta_nodal->ncomp, up_int_host.ddtheta_nodal->size);
  up_int_dev->curlbhat_nodal = gkyl_array_cu_dev_new(up_int_host.curlbhat_nodal->type,
    up_int_host.curlbhat_nodal->ncomp, up_int_host.curlbhat_nodal->size);
  up_int_dev->dualcurlbhat_nodal = gkyl_array_cu_dev_new(up_int_host.dualcurlbhat_nodal->type,
    up_int_host.dualcurlbhat_nodal->ncomp, up_int_host.dualcurlbhat_nodal->size);
  up_int_dev->jacobgeo_nodal = gkyl_array_cu_dev_new(up_int_host.jacobgeo_nodal->type,
    up_int_host.jacobgeo_nodal->ncomp, up_int_host.jacobgeo_nodal->size);
  up_int_dev->g_ij_nodal = gkyl_array_cu_dev_new(up_int_host.g_ij_nodal->type,
    up_int_host.g_ij_nodal->ncomp, up_int_host.g_ij_nodal->size);
  up_int_dev->g_ij_neut_nodal = gkyl_array_cu_dev_new(up_int_host.g_ij_neut_nodal->type,
    up_int_host.g_ij_neut_nodal->ncomp, up_int_host.g_ij_neut_nodal->size);
  up_int_dev->dxdz_nodal = gkyl_array_cu_dev_new(up_int_host.dxdz_nodal->type,
    up_int_host.dxdz_nodal->ncomp, up_int_host.dxdz_nodal->size);
  up_int_dev->dzdx_nodal = gkyl_array_cu_dev_new(up_int_host.dzdx_nodal->type,
    up_int_host.dzdx_nodal->ncomp, up_int_host.dzdx_nodal->size);
  up_int_dev->dualmag_nodal = gkyl_array_cu_dev_new(up_int_host.dualmag_nodal->type,
    up_int_host.dualmag_nodal->ncomp, up_int_host.dualmag_nodal->size);
  up_int_dev->normals_nodal = gkyl_array_cu_dev_new(up_int_host.normals_nodal->type,
    up_int_host.normals_nodal->ncomp, up_int_host.normals_nodal->size);
  up_int_dev->gij_neut_nodal = gkyl_array_cu_dev_new(up_int_host.gij_neut_nodal->type,
    up_int_host.gij_neut_nodal->ncomp, up_int_host.gij_neut_nodal->size);
  up_int_dev->b_i_nodal = gkyl_array_cu_dev_new(up_int_host.b_i_nodal->type,
    up_int_host.b_i_nodal->ncomp, up_int_host.b_i_nodal->size);
  up_int_dev->b_i_nodal_fd = gkyl_array_cu_dev_new(up_int_host.b_i_nodal_fd->type,
    up_int_host.b_i_nodal_fd->ncomp, up_int_host.b_i_nodal_fd->size);
  up_int_dev->bcart_nodal = gkyl_array_cu_dev_new(up_int_host.bcart_nodal->type,
    up_int_host.bcart_nodal->ncomp, up_int_host.bcart_nodal->size);
  up_int_dev->B3_nodal = gkyl_array_cu_dev_new(up_int_host.B3_nodal->type,
    up_int_host.B3_nodal->ncomp, up_int_host.B3_nodal->size);
  up_int_dev->dualcurlbhatoverB_nodal = gkyl_array_cu_dev_new(up_int_host.dualcurlbhatoverB_nodal->type,
    up_int_host.dualcurlbhatoverB_nodal->ncomp, up_int_host.dualcurlbhatoverB_nodal->size);
  up_int_dev->rtg33inv_nodal = gkyl_array_cu_dev_new(up_int_host.rtg33inv_nodal->type,
    up_int_host.rtg33inv_nodal->ncomp, up_int_host.rtg33inv_nodal->size);
  up_int_dev->bioverJB_nodal = gkyl_array_cu_dev_new(up_int_host.bioverJB_nodal->type,
    up_int_host.bioverJB_nodal->ncomp, up_int_host.bioverJB_nodal->size);
  return up_int_dev;
}

struct gk_geom_surf*
gk_geometry_surf_cu_dev_alloc(struct gk_geom_surf up_surf_host)
{
  struct gk_geom_surf *up_surf_dev = (struct gk_geom_surf*) gkyl_malloc(sizeof(struct gk_geom_surf));
  // Expansions.
  up_surf_dev->bmag = gkyl_array_cu_dev_new(up_surf_host.bmag->type,
    up_surf_host.bmag->ncomp, up_surf_host.bmag->size);
  up_surf_dev->jacobgeo = gkyl_array_cu_dev_new(up_surf_host.jacobgeo->type,
    up_surf_host.jacobgeo->ncomp, up_surf_host.jacobgeo->size);
  up_surf_dev->jacobgeo_sync = gkyl_array_cu_dev_new(up_surf_host.jacobgeo_sync->type,
    up_surf_host.jacobgeo_sync->ncomp, up_surf_host.jacobgeo_sync->size);
  up_surf_dev->b_i = gkyl_array_cu_dev_new(up_surf_host.b_i->type,
    up_surf_host.b_i->ncomp, up_surf_host.b_i->size);
  up_surf_dev->cmag = gkyl_array_cu_dev_new(up_surf_host.cmag->type,
    up_surf_host.cmag->ncomp, up_surf_host.cmag->size);
  up_surf_dev->jacobtot_inv = gkyl_array_cu_dev_new(up_surf_host.jacobtot_inv->type,
    up_surf_host.jacobtot_inv->ncomp, up_surf_host.jacobtot_inv->size);
  up_surf_dev->B3 = gkyl_array_cu_dev_new(up_surf_host.B3->type,
    up_surf_host.B3->ncomp, up_surf_host.B3->size);
  up_surf_dev->normcurlbhat = gkyl_array_cu_dev_new(up_surf_host.normcurlbhat->type,
    up_surf_host.normcurlbhat->ncomp, up_surf_host.normcurlbhat->size);
  up_surf_dev->normals = gkyl_array_cu_dev_new(up_surf_host.normals->type,
    up_surf_host.normals->ncomp, up_surf_host.normals->size);
  up_surf_dev->lenr = gkyl_array_cu_dev_new(up_surf_host.lenr->type,
    up_surf_host.lenr->ncomp, up_surf_host.lenr->size);
  // Nodal.
  up_surf_dev->mc2p_nodal_fd = gkyl_array_cu_dev_new(up_surf_host.mc2p_nodal_fd->type,
    up_surf_host.mc2p_nodal_fd->ncomp, up_surf_host.mc2p_nodal_fd->size);
  up_surf_dev->mc2p_nodal = gkyl_array_cu_dev_new(up_surf_host.mc2p_nodal->type,
    up_surf_host.mc2p_nodal->ncomp, up_surf_host.mc2p_nodal->size);
  up_surf_dev->bmag_nodal = gkyl_array_cu_dev_new(up_surf_host.bmag_nodal->type,
    up_surf_host.bmag_nodal->ncomp, up_surf_host.bmag_nodal->size);
  up_surf_dev->curlbhat_nodal = gkyl_array_cu_dev_new(up_surf_host.curlbhat_nodal->type,
    up_surf_host.curlbhat_nodal->ncomp, up_surf_host.curlbhat_nodal->size);
  up_surf_dev->normcurlbhat_nodal = gkyl_array_cu_dev_new(up_surf_host.normcurlbhat_nodal->type,
    up_surf_host.normcurlbhat_nodal->ncomp, up_surf_host.normcurlbhat_nodal->size);
  up_surf_dev->ddtheta_nodal = gkyl_array_cu_dev_new(up_surf_host.ddtheta_nodal->type,
    up_surf_host.ddtheta_nodal->ncomp, up_surf_host.ddtheta_nodal->size);
  up_surf_dev->jacobgeo_nodal = gkyl_array_cu_dev_new(up_surf_host.jacobgeo_nodal->type,
    up_surf_host.jacobgeo_nodal->ncomp, up_surf_host.jacobgeo_nodal->size);
  up_surf_dev->b_i_nodal = gkyl_array_cu_dev_new(up_surf_host.b_i_nodal->type,
    up_surf_host.b_i_nodal->ncomp, up_surf_host.b_i_nodal->size);
  up_surf_dev->b_i_nodal_fd = gkyl_array_cu_dev_new(up_surf_host.b_i_nodal_fd->type,
    up_surf_host.b_i_nodal_fd->ncomp, up_surf_host.b_i_nodal_fd->size);
  up_surf_dev->cmag_nodal = gkyl_array_cu_dev_new(up_surf_host.cmag_nodal->type,
    up_surf_host.cmag_nodal->ncomp, up_surf_host.cmag_nodal->size);
  up_surf_dev->jacobtot_inv_nodal = gkyl_array_cu_dev_new(up_surf_host.jacobtot_inv_nodal->type,
    up_surf_host.jacobtot_inv_nodal->ncomp, up_surf_host.jacobtot_inv_nodal->size);
  up_surf_dev->g_ij_nodal = gkyl_array_cu_dev_new(up_surf_host.g_ij_nodal->type,
    up_surf_host.g_ij_nodal->ncomp, up_surf_host.g_ij_nodal->size);
  up_surf_dev->dxdz_nodal = gkyl_array_cu_dev_new(up_surf_host.dxdz_nodal->type,
    up_surf_host.dxdz_nodal->ncomp, up_surf_host.dxdz_nodal->size);
  up_surf_dev->dzdx_nodal = gkyl_array_cu_dev_new(up_surf_host.dzdx_nodal->type,
    up_surf_host.dzdx_nodal->ncomp, up_surf_host.dzdx_nodal->size);
  up_surf_dev->normals_nodal = gkyl_array_cu_dev_new(up_surf_host.normals_nodal->type,
    up_surf_host.normals_nodal->ncomp, up_surf_host.normals_nodal->size);
  up_surf_dev->dualmag_nodal = gkyl_array_cu_dev_new(up_surf_host.dualmag_nodal->type,
    up_surf_host.dualmag_nodal->ncomp, up_surf_host.dualmag_nodal->size);
  up_surf_dev->bcart_nodal = gkyl_array_cu_dev_new(up_surf_host.bcart_nodal->type,
    up_surf_host.bcart_nodal->ncomp, up_surf_host.bcart_nodal->size);
  up_surf_dev->B3_nodal = gkyl_array_cu_dev_new(up_surf_host.B3_nodal->type,
    up_surf_host.B3_nodal->ncomp, up_surf_host.B3_nodal->size);
  up_surf_dev->lenr_nodal = gkyl_array_cu_dev_new(up_surf_host.lenr_nodal->type,
    up_surf_host.lenr_nodal->ncomp, up_surf_host.lenr_nodal->size);
  return up_surf_dev;
}

// CPU interface to create and track a GPU object
struct gk_geometry* 
gkyl_gk_geometry_cu_dev_new(struct gk_geometry* geo_host, struct gkyl_gk_geometry_inp *geometry_inp)
{
  struct gk_geometry *up =(struct gk_geometry*) gkyl_malloc(sizeof(struct gk_geometry));

  up->basis = geometry_inp->basis;
  up->local = geometry_inp->local;
  up->local_ext = geometry_inp->local_ext;
  up->global = geometry_inp->global;
  up->global_ext = geometry_inp->global_ext;
  up->grid = geometry_inp->grid;
  if (up->grid.ndim > 1) {
    gkyl_cart_modal_serendip(&up->surf_basis, up->grid.ndim-1, up->basis.poly_order);
    up->num_surf_basis = up->surf_basis.num_basis;
  }
  else {
    up->num_surf_basis = 1;
  }
  up->geqdsk_sign_convention = geo_host->geqdsk_sign_convention;
  up->has_LCFS = geo_host->has_LCFS;
  if (up->has_LCFS) {
    up->x_LCFS = geo_host->x_LCFS;
    // Check that the split happens within the domain.
    assert((up->grid.lower[0] <= up->x_LCFS) && (up->x_LCFS <= up->grid.upper[0]));
    // Check that the split happens at a cell boundary;
    double needint = (up->x_LCFS - up->grid.lower[0])/up->grid.dx[0];
    double rem_floor = fabs(needint-floor(needint));
    double rem_ceil = fabs(needint-ceil(needint));
    if (rem_floor < 1.0e-12) {
      up->idx_LCFS_lo = (int) floor(needint);
    }
    else if (rem_ceil < 1.0e-12) {
      up->idx_LCFS_lo = (int) ceil(needint);
    }
    else {
      fprintf(stderr, "x_LCFS = %.9e must be at a cell boundary.\n", up->x_LCFS);
      assert(false);
    }
  }

  struct gk_geom_corn *geo_corn_dev = gk_geometry_corn_cu_dev_alloc(geo_host->geo_corn);
  struct gk_geom_int *geo_int_dev = gk_geometry_int_cu_dev_alloc(geo_host->geo_int);
  struct gk_geom_surf *geo_surf_dev[up->grid.ndim];
  for (int dir=0; dir<up->grid.ndim; ++dir)
    geo_surf_dev[dir] = gk_geometry_surf_cu_dev_alloc(geo_host->geo_surf[dir]);

  // Expansions.
  gkyl_array_copy(geo_corn_dev->mc2p, geo_host->geo_corn.mc2p);
  gkyl_array_copy(geo_corn_dev->mc2nu_pos, geo_host->geo_corn.mc2nu_pos);
  gkyl_array_copy(geo_corn_dev->bmag, geo_host->geo_corn.bmag);
  // Nodal.
  gkyl_array_copy(geo_corn_dev->mc2p_nodal, geo_host->geo_corn.mc2p_nodal);
  gkyl_array_copy(geo_corn_dev->mc2nu_pos_nodal, geo_host->geo_corn.mc2nu_pos_nodal);
  gkyl_array_copy(geo_corn_dev->bmag_nodal, geo_host->geo_corn.bmag_nodal);

  // Expansions.
  gkyl_array_copy(geo_int_dev->mc2p, geo_host->geo_int.mc2p);
  gkyl_array_copy(geo_int_dev->bmag, geo_host->geo_int.bmag);
  gkyl_array_copy(geo_int_dev->g_ij, geo_host->geo_int.g_ij);
  gkyl_array_copy(geo_int_dev->g_ij_neut, geo_host->geo_int.g_ij_neut);
  gkyl_array_copy(geo_int_dev->dxdz, geo_host->geo_int.dxdz);
  gkyl_array_copy(geo_int_dev->dzdx, geo_host->geo_int.dzdx);
  gkyl_array_copy(geo_int_dev->dualmag, geo_host->geo_int.dualmag);
  gkyl_array_copy(geo_int_dev->normals, geo_host->geo_int.normals);
  gkyl_array_copy(geo_int_dev->jacobgeo, geo_host->geo_int.jacobgeo);
  gkyl_array_copy(geo_int_dev->jacobgeo_ghost, geo_host->geo_int.jacobgeo_ghost);
  gkyl_array_copy(geo_int_dev->jacobgeo_inv, geo_host->geo_int.jacobgeo_inv);
  gkyl_array_copy(geo_int_dev->gij, geo_host->geo_int.gij);
  gkyl_array_copy(geo_int_dev->gij_neut, geo_host->geo_int.gij_neut);
  gkyl_array_copy(geo_int_dev->b_i, geo_host->geo_int.b_i);
  gkyl_array_copy(geo_int_dev->bcart, geo_host->geo_int.bcart);
  gkyl_array_copy(geo_int_dev->cmag, geo_host->geo_int.cmag);
  gkyl_array_copy(geo_int_dev->jacobtot, geo_host->geo_int.jacobtot);
  gkyl_array_copy(geo_int_dev->jacobtot_inv, geo_host->geo_int.jacobtot_inv);
  gkyl_array_copy(geo_int_dev->bmag_inv, geo_host->geo_int.bmag_inv);
  gkyl_array_copy(geo_int_dev->bmag_inv_sq, geo_host->geo_int.bmag_inv_sq);
  gkyl_array_copy(geo_int_dev->gxxj, geo_host->geo_int.gxxj);
  gkyl_array_copy(geo_int_dev->gxyj, geo_host->geo_int.gxyj);
  gkyl_array_copy(geo_int_dev->gyyj, geo_host->geo_int.gyyj);
  gkyl_array_copy(geo_int_dev->gxzj, geo_host->geo_int.gxzj);
  gkyl_array_copy(geo_int_dev->eps2, geo_host->geo_int.eps2);
  gkyl_array_copy(geo_int_dev->rtg33inv, geo_host->geo_int.rtg33inv);
  gkyl_array_copy(geo_int_dev->dualcurlbhatoverB, geo_host->geo_int.dualcurlbhatoverB);
  gkyl_array_copy(geo_int_dev->bioverJB, geo_host->geo_int.bioverJB);
  gkyl_array_copy(geo_int_dev->B3, geo_host->geo_int.B3);
  gkyl_array_copy(geo_int_dev->dualcurlbhat, geo_host->geo_int.dualcurlbhat);
  // Nodal.
  gkyl_array_copy(geo_int_dev->mc2p_nodal_fd, geo_host->geo_int.mc2p_nodal_fd); 
  gkyl_array_copy(geo_int_dev->mc2p_nodal, geo_host->geo_int.mc2p_nodal); 
  gkyl_array_copy(geo_int_dev->bmag_nodal, geo_host->geo_int.bmag_nodal); 
  gkyl_array_copy(geo_int_dev->ddtheta_nodal, geo_host->geo_int.ddtheta_nodal); 
  gkyl_array_copy(geo_int_dev->curlbhat_nodal, geo_host->geo_int.curlbhat_nodal); 
  gkyl_array_copy(geo_int_dev->dualcurlbhat_nodal, geo_host->geo_int.dualcurlbhat_nodal); 
  gkyl_array_copy(geo_int_dev->jacobgeo_nodal, geo_host->geo_int.jacobgeo_nodal); 
  gkyl_array_copy(geo_int_dev->g_ij_nodal, geo_host->geo_int.g_ij_nodal); 
  gkyl_array_copy(geo_int_dev->g_ij_neut_nodal, geo_host->geo_int.g_ij_neut_nodal); 
  gkyl_array_copy(geo_int_dev->dxdz_nodal, geo_host->geo_int.dxdz_nodal); 
  gkyl_array_copy(geo_int_dev->dzdx_nodal, geo_host->geo_int.dzdx_nodal); 
  gkyl_array_copy(geo_int_dev->dualmag_nodal, geo_host->geo_int.dualmag_nodal); 
  gkyl_array_copy(geo_int_dev->normals_nodal, geo_host->geo_int.normals_nodal); 
  gkyl_array_copy(geo_int_dev->gij_neut_nodal, geo_host->geo_int.gij_neut_nodal); 
  gkyl_array_copy(geo_int_dev->b_i_nodal, geo_host->geo_int.b_i_nodal); 
  gkyl_array_copy(geo_int_dev->b_i_nodal_fd, geo_host->geo_int.b_i_nodal_fd); 
  gkyl_array_copy(geo_int_dev->bcart_nodal, geo_host->geo_int.bcart_nodal); 
  gkyl_array_copy(geo_int_dev->B3_nodal, geo_host->geo_int.B3_nodal); 
  gkyl_array_copy(geo_int_dev->dualcurlbhatoverB_nodal, geo_host->geo_int.dualcurlbhatoverB_nodal); 
  gkyl_array_copy(geo_int_dev->rtg33inv_nodal, geo_host->geo_int.rtg33inv_nodal); 
  gkyl_array_copy(geo_int_dev->bioverJB_nodal, geo_host->geo_int.bioverJB_nodal); 

  for (int dir=0; dir<up->grid.ndim; ++dir) {
    // Expansions.
    gkyl_array_copy(geo_surf_dev[dir]->bmag, geo_host->geo_surf[dir].bmag);
    gkyl_array_copy(geo_surf_dev[dir]->jacobgeo, geo_host->geo_surf[dir].jacobgeo);
    gkyl_array_copy(geo_surf_dev[dir]->jacobgeo_sync, geo_host->geo_surf[dir].jacobgeo_sync);
    gkyl_array_copy(geo_surf_dev[dir]->b_i, geo_host->geo_surf[dir].b_i);
    gkyl_array_copy(geo_surf_dev[dir]->cmag, geo_host->geo_surf[dir].cmag);
    gkyl_array_copy(geo_surf_dev[dir]->jacobtot_inv, geo_host->geo_surf[dir].jacobtot_inv);
    gkyl_array_copy(geo_surf_dev[dir]->B3, geo_host->geo_surf[dir].B3);
    gkyl_array_copy(geo_surf_dev[dir]->normcurlbhat, geo_host->geo_surf[dir].normcurlbhat);
    gkyl_array_copy(geo_surf_dev[dir]->normals, geo_host->geo_surf[dir].normals);
    gkyl_array_copy(geo_surf_dev[dir]->lenr, geo_host->geo_surf[dir].lenr);
    // Nodal.
    gkyl_array_copy(geo_surf_dev[dir]->mc2p_nodal_fd, geo_host->geo_surf[dir].mc2p_nodal_fd);
    gkyl_array_copy(geo_surf_dev[dir]->mc2p_nodal, geo_host->geo_surf[dir].mc2p_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->bmag_nodal, geo_host->geo_surf[dir].bmag_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->curlbhat_nodal, geo_host->geo_surf[dir].curlbhat_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->normcurlbhat_nodal, geo_host->geo_surf[dir].normcurlbhat_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->ddtheta_nodal, geo_host->geo_surf[dir].ddtheta_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->jacobgeo_nodal, geo_host->geo_surf[dir].jacobgeo_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->b_i_nodal, geo_host->geo_surf[dir].b_i_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->b_i_nodal_fd, geo_host->geo_surf[dir].b_i_nodal_fd);
    gkyl_array_copy(geo_surf_dev[dir]->cmag_nodal, geo_host->geo_surf[dir].cmag_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->jacobtot_inv_nodal, geo_host->geo_surf[dir].jacobtot_inv_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->g_ij_nodal, geo_host->geo_surf[dir].g_ij_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->dxdz_nodal, geo_host->geo_surf[dir].dxdz_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->dzdx_nodal, geo_host->geo_surf[dir].dzdx_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->normals_nodal, geo_host->geo_surf[dir].normals_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->dualmag_nodal, geo_host->geo_surf[dir].dualmag_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->bcart_nodal, geo_host->geo_surf[dir].bcart_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->B3_nodal, geo_host->geo_surf[dir].B3_nodal);
    gkyl_array_copy(geo_surf_dev[dir]->lenr_nodal, geo_host->geo_surf[dir].lenr_nodal);
  }

  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);
  up->ref_count = gkyl_ref_count_init(gkyl_gk_geometry_free);

  // Initialize the device geometry object
  struct gk_geometry *up_cu = (struct gk_geometry*) gkyl_cu_malloc(sizeof(struct gk_geometry));
  gkyl_cu_memcpy(up_cu, up, sizeof(struct gk_geometry), GKYL_CU_MEMCPY_H2D);
  gkyl_geometry_set_corn_cu(up_cu, geo_corn_dev);
  gkyl_geometry_set_int_cu(up_cu, geo_int_dev);
  for (int dir=0; dir<up->grid.ndim; ++dir)
    gkyl_geometry_set_surf_cu(up_cu, geo_surf_dev[dir], dir);

  up->on_dev = up_cu;

  // Geometry object should store host pointer.
  // Expansions.
  up->geo_corn.mc2p = geo_corn_dev->mc2p;
  up->geo_corn.mc2nu_pos = geo_corn_dev->mc2nu_pos;
  up->geo_corn.bmag = geo_corn_dev->bmag;
  up->geo_corn.mc2p_deflated = geo_corn_dev->mc2p_deflated;
  up->geo_corn.mc2nu_pos_deflated = geo_corn_dev->mc2nu_pos_deflated;
  // Nodal.
  up->geo_corn.mc2p_nodal = geo_corn_dev->mc2p_nodal;
  up->geo_corn.mc2nu_pos_nodal = geo_corn_dev->mc2nu_pos_nodal;
  up->geo_corn.bmag_nodal = geo_corn_dev->bmag_nodal;
  gkyl_free(geo_corn_dev);

  // Expansions.
  up->geo_int.mc2p = geo_int_dev->mc2p;
  up->geo_int.bmag = geo_int_dev->bmag;
  up->geo_int.g_ij = geo_int_dev->g_ij;
  up->geo_int.g_ij_neut = geo_int_dev->g_ij_neut;
  up->geo_int.dxdz = geo_int_dev->dxdz;
  up->geo_int.dzdx = geo_int_dev->dzdx;
  up->geo_int.dualmag = geo_int_dev->dualmag;
  up->geo_int.normals = geo_int_dev->normals;
  up->geo_int.jacobgeo = geo_int_dev->jacobgeo;
  up->geo_int.jacobgeo_ghost = geo_int_dev->jacobgeo_ghost;
  up->geo_int.jacobgeo_inv = geo_int_dev->jacobgeo_inv;
  up->geo_int.gij = geo_int_dev->gij;
  up->geo_int.gij_neut = geo_int_dev->gij_neut;
  up->geo_int.b_i = geo_int_dev->b_i;
  up->geo_int.bcart = geo_int_dev->bcart;
  up->geo_int.cmag = geo_int_dev->cmag;
  up->geo_int.jacobtot = geo_int_dev->jacobtot;
  up->geo_int.jacobtot_inv = geo_int_dev->jacobtot_inv;
  up->geo_int.bmag_inv = geo_int_dev->bmag_inv;
  up->geo_int.bmag_inv_sq = geo_int_dev->bmag_inv_sq;
  up->geo_int.gxxj = geo_int_dev->gxxj;
  up->geo_int.gxyj = geo_int_dev->gxyj;
  up->geo_int.gyyj = geo_int_dev->gyyj;
  up->geo_int.gxzj = geo_int_dev->gxzj;
  up->geo_int.eps2 = geo_int_dev->eps2;
  up->geo_int.rtg33inv = geo_int_dev->rtg33inv;
  up->geo_int.dualcurlbhatoverB = geo_int_dev->dualcurlbhatoverB;
  up->geo_int.bioverJB = geo_int_dev->bioverJB;
  up->geo_int.B3 = geo_int_dev->B3;
  up->geo_int.dualcurlbhat = geo_int_dev->dualcurlbhat;
  // Nodal.
  up->geo_int.mc2p_nodal_fd = geo_int_dev->mc2p_nodal_fd;
  up->geo_int.mc2p_nodal = geo_int_dev->mc2p_nodal;
  up->geo_int.bmag_nodal = geo_int_dev->bmag_nodal;
  up->geo_int.ddtheta_nodal = geo_int_dev->ddtheta_nodal;
  up->geo_int.curlbhat_nodal = geo_int_dev->curlbhat_nodal;
  up->geo_int.dualcurlbhat_nodal = geo_int_dev->dualcurlbhat_nodal;
  up->geo_int.jacobgeo_nodal = geo_int_dev->jacobgeo_nodal;
  up->geo_int.g_ij_nodal = geo_int_dev->g_ij_nodal;
  up->geo_int.g_ij_neut_nodal = geo_int_dev->g_ij_neut_nodal;
  up->geo_int.dxdz_nodal = geo_int_dev->dxdz_nodal;
  up->geo_int.dzdx_nodal = geo_int_dev->dzdx_nodal;
  up->geo_int.dualmag_nodal = geo_int_dev->dualmag_nodal;
  up->geo_int.normals_nodal = geo_int_dev->normals_nodal;
  up->geo_int.gij_neut_nodal = geo_int_dev->gij_neut_nodal;
  up->geo_int.b_i_nodal = geo_int_dev->b_i_nodal;
  up->geo_int.b_i_nodal_fd = geo_int_dev->b_i_nodal_fd;
  up->geo_int.bcart_nodal = geo_int_dev->bcart_nodal;
  up->geo_int.B3_nodal = geo_int_dev->B3_nodal;
  up->geo_int.dualcurlbhatoverB_nodal = geo_int_dev->dualcurlbhatoverB_nodal;
  up->geo_int.rtg33inv_nodal = geo_int_dev->rtg33inv_nodal;
  up->geo_int.bioverJB_nodal = geo_int_dev->bioverJB_nodal;
  gkyl_free(geo_int_dev);

  for (int dir=0; dir<up->grid.ndim; ++dir) {
    // Expansions.
    up->geo_surf[dir].bmag = geo_surf_dev[dir]->bmag;
    up->geo_surf[dir].jacobgeo = geo_surf_dev[dir]->jacobgeo;
    up->geo_surf[dir].jacobgeo_sync = geo_surf_dev[dir]->jacobgeo_sync;
    up->geo_surf[dir].b_i = geo_surf_dev[dir]->b_i;
    up->geo_surf[dir].cmag = geo_surf_dev[dir]->cmag;
    up->geo_surf[dir].jacobtot_inv = geo_surf_dev[dir]->jacobtot_inv;
    up->geo_surf[dir].B3 = geo_surf_dev[dir]->B3;
    up->geo_surf[dir].normcurlbhat = geo_surf_dev[dir]->normcurlbhat;
    up->geo_surf[dir].normals = geo_surf_dev[dir]->normals;
    up->geo_surf[dir].lenr = geo_surf_dev[dir]->lenr;
    // Nodal.
    up->geo_surf[dir].mc2p_nodal_fd = geo_surf_dev[dir]->mc2p_nodal_fd;
    up->geo_surf[dir].mc2p_nodal = geo_surf_dev[dir]->mc2p_nodal;
    up->geo_surf[dir].bmag_nodal = geo_surf_dev[dir]->bmag_nodal;
    up->geo_surf[dir].curlbhat_nodal = geo_surf_dev[dir]->curlbhat_nodal;
    up->geo_surf[dir].normcurlbhat_nodal = geo_surf_dev[dir]->normcurlbhat_nodal;
    up->geo_surf[dir].ddtheta_nodal = geo_surf_dev[dir]->ddtheta_nodal;
    up->geo_surf[dir].jacobgeo_nodal = geo_surf_dev[dir]->jacobgeo_nodal;
    up->geo_surf[dir].b_i_nodal = geo_surf_dev[dir]->b_i_nodal;
    up->geo_surf[dir].b_i_nodal_fd = geo_surf_dev[dir]->b_i_nodal_fd;
    up->geo_surf[dir].cmag_nodal = geo_surf_dev[dir]->cmag_nodal;
    up->geo_surf[dir].jacobtot_inv_nodal = geo_surf_dev[dir]->jacobtot_inv_nodal;
    up->geo_surf[dir].g_ij_nodal = geo_surf_dev[dir]->g_ij_nodal;
    up->geo_surf[dir].dxdz_nodal = geo_surf_dev[dir]->dxdz_nodal;
    up->geo_surf[dir].dzdx_nodal = geo_surf_dev[dir]->dzdx_nodal;
    up->geo_surf[dir].normals_nodal = geo_surf_dev[dir]->normals_nodal;
    up->geo_surf[dir].dualmag_nodal = geo_surf_dev[dir]->dualmag_nodal;
    up->geo_surf[dir].bcart_nodal = geo_surf_dev[dir]->bcart_nodal;
    up->geo_surf[dir].B3_nodal = geo_surf_dev[dir]->B3_nodal;
    up->geo_surf[dir].lenr_nodal = geo_surf_dev[dir]->lenr_nodal;
    gkyl_free(geo_surf_dev[dir]);
  }
 
  return up;
}

